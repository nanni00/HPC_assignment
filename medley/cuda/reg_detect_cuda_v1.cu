#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is int, default size is 50. */
#include "reg_detect.h"


#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

static void init_array(int maxgrid, int* sum_tang, int* mean, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      sum_tang[i * maxgrid + j] = (int)((i + 1) * (j + 1));
      mean[i * maxgrid + j] = ((int)i - j) / maxgrid;
      path[i * maxgrid + j] = ((int)i * (j - 1)) / maxgrid;
    }
}

static void print_array(int maxgrid, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      fprintf(stdout, "%d ", path[i * maxgrid + j]);
      if ((i * maxgrid + j) % 20 == 0)
        fprintf(stdout, "\n");
    }
  fprintf(stdout, "\n");
}

__global__ void kernel(int maxgrid, int length,int* sum_tang_d,int* mean_d, int* path_d, int* diff_d, int* sum_diff_d, int selector) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= maxgrid || j >= maxgrid)
    return;
    
  if (selector == 1) {
    //Primo for
    for (int cnt = 0; cnt <= length - 1; cnt++)
      diff_d[(j*maxgrid + i)*length + cnt] = sum_tang_d[j*maxgrid + i];  
      
  } else if (selector == 2) {
    //Secondo for
    sum_diff_d[(j*maxgrid + i)*length] = diff_d[(j*maxgrid + i)*length];

    for (int cnt = 1; cnt <= length - 1; cnt++)
      sum_diff_d[(j*maxgrid + i)*length + cnt] = sum_diff_d[(j*maxgrid + i)*length + cnt - 1] + diff_d[(j*maxgrid + i)*length + cnt];
    mean_d[j*maxgrid + i] = sum_diff_d[(j*maxgrid + i)*length + (length - 1)];
  
  } else if (selector == 3 && j == 0) {
    //Terzo for
    path_d[i] = mean_d[i];

  } else if (selector == 4 && i == 0 && j == 0) {
    //Quarto for
    for (j = 1; j <= maxgrid - 1; j++)
      for (i = j; i <= maxgrid - 1; i++)
        path_d[j*maxgrid + i] = path_d[(j - 1)*maxgrid + (i-1)] + mean_d[j*maxgrid + i];
  }
}

#define BLOCK_SIZE 32

static void kernel_reg_detect(int niter, int maxgrid, int length,int* sum_tang_d,int* mean_d,
                              int* path_d, int* diff_d, int* sum_diff_d)
{

  //dim3 dimGrid((maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE,(maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE);
  //dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);

  dim3 dimGrid(1, 1);
  dim3 dimBlock(maxgrid,maxgrid);

  clock_t begin = clock();
  
  for (int t = 0; t < niter; t++)
  { 
    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 1);
    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 2);
    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 3);
    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 4);
  }

  clock_t end = clock();
  printf("Elapsed time with custom timer: %lf\n", (double)(end - begin) / CLOCKS_PER_SEC);
}

/**
 * Tutto viene eleborato sulla GPU e poi copiato su host.
*/
int main() {
    int niter = NITER;
    int maxgrid = MAXGRID;
    int length = LENGTH;

    int* sum_tang = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    //int* sum_diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    //int* diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    int* mean = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* path = (int*)malloc(sizeof(int) * maxgrid * maxgrid);

    int* sum_tang_d;
    int* sum_diff_d;
    int* mean_d;
    int* path_d;
    int* diff_d;
    
    // allocation on device
    hipMalloc((void**)&sum_tang_d, sizeof(int) * maxgrid * maxgrid);
    hipMalloc((void**)&sum_diff_d, sizeof(int) * maxgrid * maxgrid * length);
    hipMalloc((void**)&mean_d, sizeof(int) * maxgrid * maxgrid);
    hipMalloc((void**)&path_d, sizeof(int) * maxgrid * maxgrid);
    hipMalloc((void**)&diff_d, sizeof(int) * maxgrid * maxgrid * length);

    init_array(maxgrid, sum_tang, mean, path);
    
    print_array(maxgrid, path);

    // copies to the device
    hipMemcpy(sum_tang_d, sum_tang, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(mean_d, mean, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(path_d, path, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);

    // call the function
    kernel_reg_detect(niter, maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d);

    // copy to the host
    hipMemcpy(path, path_d, sizeof(int) * maxgrid * maxgrid, hipMemcpyDeviceToHost);

    print_array(maxgrid, path);

    // free host memory
    free(sum_tang);
    free(mean);
    free(path);

    hipFree(sum_tang_d);
    hipFree(mean_d);
    hipFree(path_d);
    hipFree(diff_d);
    hipFree(sum_diff_d);

    return 0;
}
