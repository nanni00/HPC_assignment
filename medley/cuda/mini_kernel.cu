#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is int, default size is 50. */
#include "reg_detect.h"

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

static void init_array(int maxgrid, int* sum_tang, int* mean, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      sum_tang[i * maxgrid + j] = (int)((i + 1) * (j + 1));
      mean[i * maxgrid + j] = ((int)i - j) / maxgrid;
      path[i * maxgrid + j] = ((int)i * (j - 1)) / maxgrid;
    }
}

static void print_array(int maxgrid, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      fprintf(stdout, "%d ", path[i * maxgrid + j]);
      if ((i * maxgrid + j) % 20 == 0)
        fprintf(stdout, "\n");
    }
  fprintf(stdout, "\n");
}

__device__ void kernel_1(int maxgrid, int length, int* diff, int* sum_tang)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i <= maxgrid-1 && j <= maxgrid-1)
    {
        for (int cnt = 0; cnt <= length - 1; cnt++)
          diff[(j*maxgrid + i)*length + cnt] = sum_tang[j*maxgrid + i];
    }
}

__device__ void kernel_2(int maxgrid, int length, int* sum_diff, int*diff)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i <= maxgrid-1 && j <= maxgrid-1)
    {
        sum_diff[(j*maxgrid + i)*length] = diff[(j*maxgrid + i)*length];
    }
}

__device__ void kernel_3(int maxgrid, int length, int* sum_diff, int* diff)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i <= maxgrid-1 && j <= maxgrid-1)
    {
        for (int cnt = 1; cnt <= length - 1; cnt++)
        {
            sum_diff[(j*maxgrid + i)*length + cnt] = sum_diff[(j*maxgrid + i)*length + cnt - 1] + diff[(j*maxgrid + i)*length + cnt];
        }
    }
}

__device__ void kernel_4(int maxgrid, int length, int* mean, int* sum_diff)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i <= maxgrid-1 && j <= maxgrid-1)
    {
        mean[j*maxgrid + i] = sum_diff[(j*maxgrid + i)*length + (length - 1)];
    }
}

__device__ void kernel_5(int maxgrid, int* path, int* mean)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<=maxgrid-1)
    {
        path[i] = mean[i];
    }
}

__device__ void kernel_6(int maxgrid, int* path, int* mean)
{
    for (int j = 1; j <= maxgrid - 1; j++){
      for (int i = j; i <= maxgrid - 1; i++)
      {
        path[j*maxgrid + i] = path[(j - 1)*maxgrid + (i-1)] + mean[j*maxgrid + i];
      }
    }
}

__global__ void main_kernel(int niter, int maxgrid, int length,int* sum_tang_d,int* mean_d,
                              int* path_d, int* diff_d, int* sum_diff_d)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
        if(i<niter)
        {
            kernel_1(maxgrid, length, diff_d, sum_tang_d);
            __syncthreads();
            kernel_2(maxgrid, length, sum_diff_d, diff_d);
            __syncthreads();
            kernel_3(maxgrid, length, sum_diff_d, diff_d);
            __syncthreads();
            kernel_4(maxgrid, length, mean_d, sum_diff_d);
            __syncthreads();
            kernel_5(maxgrid, path_d, mean_d);
            __syncthreads();
            kernel_6(maxgrid, path_d, mean_d);
            __syncthreads();
        }
        //printf("i: %d\n", i);
}

void reg_detect(int niter, int maxgrid, int length,int* sum_tang,int*mean,
                              int* path, int* diff, int* sum_diff)
{
    //Allocazione array su device
    int* sum_tang_d;
    hipMalloc((void**)&sum_tang_d, sizeof(int) * maxgrid * maxgrid);

    int* mean_d;
    hipMalloc((void**)&mean_d, sizeof(int) * maxgrid * maxgrid);
    
    int* path_d;
    hipMalloc((void**)&path_d, sizeof(int) * maxgrid * maxgrid);

    int* diff_d;
    hipMalloc((void**)&diff_d, sizeof(int) * maxgrid * maxgrid * length);

    int* sum_diff_d;
    hipMalloc((void**)&sum_diff_d, sizeof(int) * maxgrid * maxgrid * length);

    //copia dati su GPU
    hipMemcpy(sum_tang_d, sum_tang, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(mean_d, mean, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(path_d, path, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(diff_d, diff, sizeof(int) * maxgrid * maxgrid * length, hipMemcpyHostToDevice);
    hipMemcpy(sum_diff_d, sum_diff, sizeof(int) * maxgrid * maxgrid * length, hipMemcpyHostToDevice);

    dim3 BlocksDim (1,1);
    dim3 ThreadsPerBlock ( maxgrid, maxgrid);
    //dim3 ThreadsPerBlock(max, 32);
	//dim3 BlocksDim((unsigned int)ceil( ((float)maxgrid) / ((float)block.x) ), (unsigned int)ceil( ((float)maxgrid) / ((float)block.y) ));

    clock_t begin = clock();

    //________________________________Scrivere chiamate ai kernel_____________________________
    main_kernel<<<BlocksDim,ThreadsPerBlock>>>(niter, maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d);
    
    clock_t end = clock();
    printf("Elapsed time with custom timer: %lf\n", (double)(end - begin) / CLOCKS_PER_SEC);

    hipMemcpy(path, path_d, sizeof(int) * maxgrid * maxgrid, hipMemcpyDeviceToHost);

    hipFree(sum_tang_d);
    hipFree(mean_d);
    hipFree(path_d);
    hipFree(diff_d);
    hipFree(sum_diff_d);


}


int main() {
    int niter = NITER;
    int maxgrid = MAXGRID;
    int length = LENGTH;

    //Allocazione array su host
    int* sum_tang = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* mean = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* path = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    int* sum_diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    
    init_array(maxgrid, sum_tang, mean, path);
    print_array(maxgrid, path);

    reg_detect(niter, maxgrid, length, sum_tang, mean, path, diff, sum_diff);

    print_array(maxgrid, path);



    free(sum_tang);
    free(mean);
    free(path);


    
    
}