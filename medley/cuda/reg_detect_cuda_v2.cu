#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is int, default size is 50. */
#include "reg_detect.h"


#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

static void init_array(int maxgrid, int* sum_tang, int* mean, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      sum_tang[i * maxgrid + j] = (int)((i + 1) * (j + 1));
      mean[i * maxgrid + j] = ((int)i - j) / maxgrid;
      path[i * maxgrid + j] = ((int)i * (j - 1)) / maxgrid;
    }
}

static void print_array(int maxgrid, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      fprintf(stdout, "%d ", path[i * maxgrid + j]);
      if ((i * maxgrid + j) % 20 == 0)
        fprintf(stdout, "\n");
    }
  fprintf(stdout, "\n");
}

/*
static void kernel_reg_detect(int niter, int maxgrid, int length,int* sum_tang,int* mean,
                              int* path, int* diff, int* sum_diff)
{
  clock_t begin = clock();

  for (int t = 0; t < niter; t++)
  { 
    for (j = 0; j <= maxgrid - 1; j++)
      for (i = j; i <= maxgrid - 1; i++)
        for (cnt = 0; cnt <= length - 1; cnt++)
          diff[(j*maxgrid + i)*length + cnt] = sum_tang[j*maxgrid + i];  

    for (j = 0; j <= maxgrid - 1; j++)
    {
      for (i = j; i <= maxgrid - 1; i++)
      {
        sum_diff[(j*maxgrid + i)*length] = diff[(j*maxgrid + i)*length];

        for (cnt = 1; cnt <= length - 1; cnt++)
          sum_diff[(j*maxgrid + i)*length + cnt] = sum_diff[(j*maxgrid + i)*length + cnt - 1] + diff[(j*maxgrid + i)*length + cnt];
        mean[j*maxgrid + i] = sum_diff[(j*maxgrid + i)*length + (length - 1)];
      }
    }

    for (i = 0; i <= maxgrid - 1; i++)
      path[i] = mean[i];

    for (j = 1; j <= maxgrid - 1; j++)
      for (i = j; i <= maxgrid - 1; i++)
        path[j*maxgrid + i] = path[(j - 1)*maxgrid + (i-1)] + mean[j*maxgrid + i];
  }

  clock_t end = clock();
  printf("Elapsed time with custom timer: %lf\n", (double)(end - begin) / CLOCKS_PER_SEC);
}
*/

__global__ void kernel(int maxgrid, int length,int* sum_tang_d,int* mean_d, int* path_d, int* diff_d, int* sum_diff_d, int selector) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= maxgrid || j >= maxgrid)
    return;
    
  if (selector == 1) {
    //Primo for
    for (int cnt = 0; cnt <= length - 1; cnt++)
      diff_d[(j*maxgrid + i)*length + cnt] = sum_tang_d[j*maxgrid + i];  
  } else if (selector == 2) {
    //Secondo for
    sum_diff_d[(j*maxgrid + i)*length] = diff_d[(j*maxgrid + i)*length];

    for (int cnt = 1; cnt <= length - 1; cnt++)
      sum_diff_d[(j*maxgrid + i)*length + cnt] = sum_diff_d[(j*maxgrid + i)*length + cnt - 1] + diff_d[(j*maxgrid + i)*length + cnt];
    mean_d[j*maxgrid + i] = sum_diff_d[(j*maxgrid + i)*length + (length - 1)];
  } else if (selector == 3 && j == 0) {
    //Terzo for
    path_d[i] = mean_d[i];
  } else if (selector == 4 && i == 0 && j == 0) {
    //Quarto for
    for (j = 1; j <= maxgrid - 1; j++)
      for (i = j; i <= maxgrid - 1; i++)
        path_d[j*maxgrid + i] = path_d[(j - 1)*maxgrid + (i-1)] + mean_d[j*maxgrid + i];
  }
}

#define BLOCK_SIZE 32

static void kernel_reg_detect_cuda(int niter, int maxgrid, int length,int* sum_tang_d,int* mean_d,
                              int* path_d, int* diff_d, int* sum_diff_d)
{
  clock_t begin = clock();

  dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
  dim3 dimGrid((maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE,(maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE);

  //dim3 dimBlock(maxgrid,maxgrid);
  //dim3 dimGrid(1, 1);

  // Allocazioni in memoria
  for (int t = 0; t < niter; t++)
  { 
    
    int a = 0;

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 1);

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 2);

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 3);

    hipDeviceSynchronize();
    for (int j = 1; j <= maxgrid - 1; j++)
      for (int i = j; i <= maxgrid - 1; i++)
        path_d[j*maxgrid + i] = path_d[(j - 1)*maxgrid + (i-1)] + mean_d[j*maxgrid + i];
   
  }

  clock_t end = clock();
  printf("Elapsed time with custom timer: %lf\n", (double)(end - begin) / CLOCKS_PER_SEC);
}

/*

  Modalità UVM

*/
int main() {
    int niter = NITER;
    int maxgrid = MAXGRID;
    int length = LENGTH;

    int* sum_tang = (int*)malloc(sizeof(int) * maxgrid * maxgrid);


    //-------------------------------------------------------------------

    //int* diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);

    //int* sum_diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    //-------------------------------------------------------------------


    int* sum_tang_d;
    hipMalloc((void**)&sum_tang_d, sizeof(int) * maxgrid * maxgrid);

    int* mean_d_uva;
    hipMallocManaged((void**)&mean_d_uva, sizeof(int) * maxgrid * maxgrid);
    
    int* path_d_uva;
    hipMallocManaged(&path_d_uva, sizeof(int) * maxgrid * maxgrid);

    int* diff_d;
    hipMalloc((void**)&diff_d, sizeof(int) * maxgrid * maxgrid * length);

    int* sum_diff_d;
    hipMalloc((void**)&sum_diff_d, sizeof(int) * maxgrid * maxgrid * length);
    
    init_array(maxgrid, sum_tang, mean_d_uva, path_d_uva);

    // Copia su GPU
    hipMemcpy(sum_tang_d, sum_tang, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);

    //print_array(maxgrid, path_d_uva);
    
    //kernel_reg_detect(niter, maxgrid, length, sum_tang, mean, path, diff, sum_diff);

    //kernel_reg_detect(niter, maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d);

    kernel_reg_detect_cuda(niter, maxgrid, length, sum_tang_d, mean_d_uva, path_d_uva, diff_d, sum_diff_d);
    hipDeviceSynchronize();


    //print_array(maxgrid, path_d_uva);



    free(sum_tang);

    hipFree(sum_tang_d);
    hipFree(mean_d_uva);
    hipFree(path_d_uva);
    hipFree(diff_d);
    hipFree(sum_diff_d);
    
    
}
