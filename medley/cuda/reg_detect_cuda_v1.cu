#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is int, default size is 50. */
#include "reg_detect.h"


#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

static void init_array(int maxgrid, int* sum_tang, int* mean, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      sum_tang[i * maxgrid + j] = (int)((i + 1) * (j + 1));
      mean[i * maxgrid + j] = ((int)i - j) / maxgrid;
      path[i * maxgrid + j] = ((int)i * (j - 1)) / maxgrid;
    }
}

static void print_array(int maxgrid, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      fprintf(stdout, "%d ", path[i * maxgrid + j]);
      if ((i * maxgrid + j) % 20 == 0)
        fprintf(stdout, "\n");
    }
  fprintf(stdout, "\n");
}

__global__ void kernel(int maxgrid, int length,int* sum_tang_d,int* mean_d, int* path_d, int* diff_d, int* sum_diff_d, int selector) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= maxgrid || j >= maxgrid)
    return;
    
  if (selector == 1) {
    //Primo for
    for (int cnt = 0; cnt <= length - 1; cnt++)
      diff_d[(j*maxgrid + i)*length + cnt] = sum_tang_d[j*maxgrid + i];  
  } else if (selector == 2) {
    //Secondo for
    sum_diff_d[(j*maxgrid + i)*length] = diff_uva[(j*maxgrid + i)*length];

    for (int cnt = 1; cnt <= length - 1; cnt++)
      sum_diff_d[(j*maxgrid + i)*length + cnt] = sum_diff_d[(j*maxgrid + i)*length + cnt - 1] + diff_d[(j*maxgrid + i)*length + cnt];
    mean_d[j*maxgrid + i] = sum_diff_uva[(j*maxgrid + i)*length + (length - 1)];
  } else if (selector == 3 && j == 0) {
    //Terzo for
    path_d[i] = mean_d[i];
  } else if (selector == 4 && i == 0 && j == 0) {
    //Quarto for
    for (j = 1; j <= maxgrid - 1; j++)
      for (i = j; i <= maxgrid - 1; i++)
        path_d[j*maxgrid + i] = path_d[(j - 1)*maxgrid + (i-1)] + mean_d[j*maxgrid + i];
  }
}

#define BLOCK_SIZE 32

static void kernel_reg_detect_cuda(int niter, int maxgrid, int length,int* sum_tang_d,int* mean_d,
                              int* path_d, int* diff_d, int* sum_diff_d)
{
  clock_t begin = clock();

  //dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
  //dim3 dimGrid((maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE,(maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE);

  dim3 dimBlock(maxgrid,maxgrid);
  dim3 dimGrid(1, 1);

  // Allocazioni in memoria
  for (int t = 0; t < niter; t++)
  { 
    
    int a = 0;

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 1);

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 2);

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 3);

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 4);

  }

  clock_t end = clock();
  printf("Elapsed time with custom timer: %lf\n", (double)(end - begin) / CLOCKS_PER_SEC);
}



/*

  Tutto viene eleborato sulla GPU e poi copiato su host.
  NB. Ciclo lento su GPU

*/
int main() {
    int niter = NITER;
    int maxgrid = MAXGRID;
    int length = LENGTH;

    int* sum_tang = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* mean = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* path = (int*)malloc(sizeof(int) * maxgrid * maxgrid);


    //-------------------------------------------------------------------

    //int* diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);

    //int* sum_diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    //-------------------------------------------------------------------


    int* sum_tang_uva;
    hipMalloc((void**)&sum_tang_uva, sizeof(int) * maxgrid * maxgrid);

    int* mean_d;
    hipMalloc((void**)&mean_d, sizeof(int) * maxgrid * maxgrid);
    
    int* path_d;
    hipMalloc((void**)&path_d, sizeof(int) * maxgrid * maxgrid);

    int* diff_uva;
    hipMalloc((void**)&diff_uva, sizeof(int) * maxgrid * maxgrid * length);

    int* sum_diff_uva;
    hipMalloc((void**)&sum_diff_uva, sizeof(int) * maxgrid * maxgrid * length);
    
    init_array(maxgrid, sum_tang, mean, path);

    // Copia su GPU
    hipMemcpy(sum_tang_uva, sum_tang, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(mean_d, mean, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(path_d, path, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);

    print_array(maxgrid, path);
    
    kernel_reg_detect_cuda(niter, maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d);

    //Copia su Host
    hipMemcpy(path, path_d, sizeof(int) * maxgrid * maxgrid, hipMemcpyDeviceToHost);

    print_array(maxgrid, path);

    free(sum_tang);
    free(mean);
    free(path);

    hipFree(sum_tang_uva);
    hipFree(mean_d);
    hipFree(path_d);
    hipFree(diff_uva);
    hipFree(sum_diff_uva);
    
    return 0;
}
