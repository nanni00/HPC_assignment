#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is int, default size is 50. */
#include "reg_detect.h"

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

static void init_array(int maxgrid, int* sum_tang, int* mean, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      sum_tang[i * maxgrid + j] = (int)((i + 1) * (j + 1));
      mean[i * maxgrid + j] = ((int)i - j) / maxgrid;
      path[i * maxgrid + j] = ((int)i * (j - 1)) / maxgrid;
    }
}

static void print_array(int maxgrid, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      fprintf(stdout, "%d ", path[i * maxgrid + j]);
      if ((i * maxgrid + j) % 20 == 0)
        fprintf(stdout, "\n");
    }
  fprintf(stdout, "\n");
}

__global__ void kernel_1(int maxgrid, int length, int* diff, int* sum_tang)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int cnt = blockIdx.z * blockDim.z + threadIdx.z; 
    //&& cnt <= length-1
    //printf("Iterazioni: %d\n", i);
    if(i <= maxgrid-1 && j <= maxgrid-1  && cnt <= length-1 && i>=j)
    {
        //for (int cnt = 0; cnt <= length - 1; cnt++)
          diff[(j*maxgrid + i)*length + cnt] = sum_tang[j*maxgrid + i];
          //printf("prova: %d\n", cnt );
    }
}

__global__ void kernel_2(int maxgrid, int length, int* sum_diff, int*diff)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i <= maxgrid-1 && j <= maxgrid-1)
    {
        sum_diff[(j*maxgrid + i)*length] = diff[(j*maxgrid + i)*length];
    }
}

__global__ void kernel_3(int maxgrid, int length, int* sum_diff, int* diff)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int cnt = blockIdx.z * blockDim.z + threadIdx.z; 

    if(i <= maxgrid-1 && j <= maxgrid-1  && cnt <= length-1 && i>=j)
    {
        //for (int cnt = 1; cnt <= length - 1; cnt++)
        //{
            sum_diff[(j*maxgrid + i)*length + cnt] = sum_diff[(j*maxgrid + i)*length + cnt - 1] + diff[(j*maxgrid + i)*length + cnt];
        //}
    }
}

__global__ void kernel_4(int maxgrid, int length, int* mean, int* sum_diff)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i <= maxgrid-1 && j <= maxgrid-1)
    {
        mean[j*maxgrid + i] = sum_diff[(j*maxgrid + i)*length + (length - 1)];
    }
}

__global__ void kernel_5(int maxgrid, int* path, int* mean)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<=maxgrid-1)
    {
        path[i] = mean[i];
    }
}

__global__ void kernel_6(int maxgrid, int* path, int* mean)
{
    for (int j = 1; j <= maxgrid - 1; j++){
      for (int i = j; i <= maxgrid - 1; i++)
      {
        path[j*maxgrid + i] = path[(j - 1)*maxgrid + (i-1)] + mean[j*maxgrid + i];
      }
    }
}


void reg_detect(int niter, int maxgrid, int length,int* sum_tang,int*mean,
                              int* path, int* diff, int* sum_diff)
{
    //Allocazione array su device
    int* sum_tang_d;
    hipMalloc((void**)&sum_tang_d, sizeof(int) * maxgrid * maxgrid);

    int* mean_d;
    hipMalloc((void**)&mean_d, sizeof(int) * maxgrid * maxgrid);
    
    int* path_d;
    hipMalloc((void**)&path_d, sizeof(int) * maxgrid * maxgrid);

    int* diff_d;
    hipMalloc((void**)&diff_d, sizeof(int) * maxgrid * maxgrid * length);

    int* sum_diff_d;
    hipMalloc((void**)&sum_diff_d, sizeof(int) * maxgrid * maxgrid * length);

    //copia dati su GPU
    hipMemcpy(sum_tang_d, sum_tang, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(mean_d, mean, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(path_d, path, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);
    hipMemcpy(diff_d, diff, sizeof(int) * maxgrid * maxgrid * length, hipMemcpyHostToDevice);
    hipMemcpy(sum_diff_d, sum_diff, sizeof(int) * maxgrid * maxgrid * length, hipMemcpyHostToDevice);
    
    //dim3 BlocksDim ((size_t)ceil(((float)niter) / ((float)maxgrid)), 1);
    
    dim3 BlocksDim (1,1);
    dim3 ThreadsPerBlock (maxgrid, maxgrid);
    dim3 BlocksDim_1 (1,1,length);
    dim3 ThreadsPerBlock_1 (maxgrid, maxgrid, 1);
    int somma = 0; 

    clock_t begin = clock();
    //printf("niter: %d\n", niter );
    for(int t = 0; t<niter; t++)
    {
    
      kernel_1<<<BlocksDim_1,ThreadsPerBlock_1>>>(maxgrid, length, diff_d, sum_tang_d);
      
      kernel_2<<<BlocksDim,ThreadsPerBlock>>>(maxgrid, length, sum_diff_d, diff_d);
      
      kernel_3<<<BlocksDim_1,ThreadsPerBlock_1>>>(maxgrid, length, sum_diff_d, diff_d);
      
      kernel_4<<<BlocksDim,ThreadsPerBlock>>>(maxgrid, length, mean_d, sum_diff_d);
      
      kernel_5<<<BlocksDim,ThreadsPerBlock>>>(maxgrid, path_d, mean_d);
      
      kernel_6<<<BlocksDim,ThreadsPerBlock>>>(maxgrid, path_d, mean_d);

      //somma++;
    }


    
    clock_t end = clock();
    printf("Elapsed time with custom timer: %lf\n", (double)(end - begin) / CLOCKS_PER_SEC);
    printf("Iterazioni: %d\n", somma);
    hipMemcpy(path, path_d, sizeof(int) * maxgrid * maxgrid, hipMemcpyDeviceToHost);

    hipFree(sum_tang_d);
    hipFree(mean_d);
    hipFree(path_d);
    hipFree(diff_d);
    hipFree(sum_diff_d);


}


int main() {
    int niter = NITER;
    int maxgrid = MAXGRID;
    int length = LENGTH;

    //Allocazione array su host
    int* sum_tang = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* mean = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* path = (int*)malloc(sizeof(int) * maxgrid * maxgrid);
    int* diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    int* sum_diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    
    init_array(maxgrid, sum_tang, mean, path);
    print_array(maxgrid, path);

    reg_detect(niter, maxgrid, length, sum_tang, mean, path, diff, sum_diff);

    print_array(maxgrid, path);



    free(sum_tang);
    free(mean);
    free(path);


    
    
}