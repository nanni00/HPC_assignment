#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is int, default size is 50. */
#include "reg_detect.h"


#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

static void init_array(int maxgrid, int* sum_tang, int* mean, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      sum_tang[i * maxgrid + j] = (int)((i + 1) * (j + 1));
      mean[i * maxgrid + j] = ((int)i - j) / maxgrid;
      path[i * maxgrid + j] = ((int)i * (j - 1)) / maxgrid;
    }
}

static void print_array(int maxgrid, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      fprintf(stdout, "%d ", path[i * maxgrid + j]);
      if ((i * maxgrid + j) % 20 == 0)
        fprintf(stdout, "\n");
    }
  fprintf(stdout, "\n");
}

__global__ void kernel_1(int maxgrid, int length, int* sum_tang_d, int* diff_d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int cnt = blockIdx.z * blockDim.z + threadIdx.z;

  if (i >= maxgrid || j >= maxgrid || cnt >= length)
    return;

  diff_d[(j*maxgrid + i)*length + cnt] = sum_tang_d[j*maxgrid + i];  
}

__global__ void kernel_2(int maxgrid, int length, int* sum_tang_d, int* diff_d) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int cnt = blockIdx.z * blockDim.z + threadIdx.z;

  if (i >= maxgrid || j >= maxgrid || cnt >= length)
    return;

  if (cnt == 0)
    sum_diff_d[(j*maxgrid + i)*length] = diff_d[(j*maxgrid + i)*length];

  for (int cnt = 1; cnt <= length - 1; cnt++)
    sum_diff_d[(j*maxgrid + i)*length + cnt] = sum_diff_d[(j*maxgrid + i)*length + cnt - 1] + diff_d[(j*maxgrid + i)*length + cnt];
  mean_d[j*maxgrid + i] = sum_diff_d[(j*maxgrid + i)*length + (length - 1)];
}

__global__ void kernel(int maxgrid, int length,int* sum_tang_d,int* mean_d, int* path_d, int* diff_d, int* sum_diff_d, int selector) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= maxgrid || j >= maxgrid)
    return;
    
  if (selector == 1) {
    //Primo for
    for (int cnt = 0; cnt <= length - 1; cnt++)
      diff_d[(j*maxgrid + i)*length + cnt] = sum_tang_d[j*maxgrid + i];  
  } else if (selector == 2) {
    //Secondo for
    sum_diff_d[(j*maxgrid + i)*length] = diff_d[(j*maxgrid + i)*length];

    for (int cnt = 1; cnt <= length - 1; cnt++)
      sum_diff_d[(j*maxgrid + i)*length + cnt] = sum_diff_d[(j*maxgrid + i)*length + cnt - 1] + diff_d[(j*maxgrid + i)*length + cnt];
    mean_d[j*maxgrid + i] = sum_diff_d[(j*maxgrid + i)*length + (length - 1)];
  } else if (selector == 3 && j == 0) {
    //Terzo for
    path_d[i] = mean_d[i];
  } else if (selector == 4 && i == 0 && j == 0) {
    //Quarto for
    for (j = 1; j <= maxgrid - 1; j++)
      for (i = j; i <= maxgrid - 1; i++)
        path_d[j*maxgrid + i] = path_d[(j - 1)*maxgrid + (i-1)] + mean_d[j*maxgrid + i];
  }
}

#define BLOCK_SIZE 32

static void kernel_reg_detect_cuda(int niter, int maxgrid, int length,int* sum_tang_d,int* mean_d,
                              int* path_d, int* diff_d, int* sum_diff_d)
{
  clock_t begin = clock();

  dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
  dim3 dimGrid((maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE,(maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE);

  //dim3 dimBlock(maxgrid,maxgrid);
  //dim3 dimGrid(1, 1);

  // Allocazioni in memoria
  for (int t = 0; t < niter; t++)
  { 
    
    int a = 0;

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 1);

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 2);

    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 3);

    hipDeviceSynchronize();
    for (int j = 1; j <= maxgrid - 1; j++)
      for (int i = j; i <= maxgrid - 1; i++)
        path_d[j*maxgrid + i] = path_d[(j - 1)*maxgrid + (i-1)] + mean_d[j*maxgrid + i];
   
  }

  clock_t end = clock();
  printf("Elapsed time with custom timer: %lf\n", (double)(end - begin) / CLOCKS_PER_SEC);
}

/*

  Modalità UVM

*/
int main() {
    int niter = NITER;
    int maxgrid = MAXGRID;
    int length = LENGTH;

    int* sum_tang = (int*)malloc(sizeof(int) * maxgrid * maxgrid);


    //-------------------------------------------------------------------

    //int* diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);

    //int* sum_diff = (int*)malloc(sizeof(int) * maxgrid * maxgrid * length);
    //-------------------------------------------------------------------


    int* sum_tang_d;
    hipMalloc((void**)&sum_tang_d, sizeof(int) * maxgrid * maxgrid);

    int* mean_d_uva;
    hipMallocManaged((void**)&mean_d_uva, sizeof(int) * maxgrid * maxgrid);
    
    int* path_d_uva;
    hipMallocManaged(&path_d_uva, sizeof(int) * maxgrid * maxgrid);

    int* diff_d;
    hipMalloc((void**)&diff_d, sizeof(int) * maxgrid * maxgrid * length);

    int* sum_diff_d;
    hipMalloc((void**)&sum_diff_d, sizeof(int) * maxgrid * maxgrid * length);
    
    init_array(maxgrid, sum_tang, mean_d_uva, path_d_uva);

    // Copia su GPU
    hipMemcpy(sum_tang_d, sum_tang, sizeof(int) * maxgrid * maxgrid, hipMemcpyHostToDevice);

    //print_array(maxgrid, path_d_uva);
    
    //kernel_reg_detect(niter, maxgrid, length, sum_tang, mean, path, diff, sum_diff);

    //kernel_reg_detect(niter, maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d);

    kernel_reg_detect_cuda(niter, maxgrid, length, sum_tang_d, mean_d_uva, path_d_uva, diff_d, sum_diff_d);
    hipDeviceSynchronize();


    //print_array(maxgrid, path_d_uva);



    free(sum_tang);

    hipFree(sum_tang_d);
    hipFree(mean_d_uva);
    hipFree(path_d_uva);
    hipFree(diff_d);
    hipFree(sum_diff_d);
    
    
}
