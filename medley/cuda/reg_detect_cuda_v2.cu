#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* Include benchmark-specific header. */
/* Default data type is int, default size is 50. */
#include "reg_detect.h"


#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

static void init_array(int maxgrid, int* sum_tang, int* mean, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      sum_tang[i * maxgrid + j] = (int)((i + 1) * (j + 1));
      mean[i * maxgrid + j] = ((int)i - j) / maxgrid;
      path[i * maxgrid + j] = ((int)i * (j - 1)) / maxgrid;
    }
}

static void print_array(int maxgrid, int* path)
{
  int i, j;

  for (i = 0; i < maxgrid; i++)
    for (j = 0; j < maxgrid; j++)
    {
      fprintf(stdout, "%d ", path[i * maxgrid + j]);
      if ((i * maxgrid + j) % 20 == 0)
        fprintf(stdout, "\n");
    }
  fprintf(stdout, "\n");
}


__global__ void kernel(int maxgrid, int length, 
                      int* sum_tang_uva,int* mean_uva, int* path_uva,
                      int* diff_uva, int* sum_diff_uva, int selector) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= maxgrid || j >= maxgrid)
    return;
    
  if (selector == 1) {
    //Primo for
    for (int cnt = 0; cnt <= length - 1; cnt++)
      diff_uva[(j*maxgrid + i)*length + cnt] = sum_tang_uva[j*maxgrid + i];  

  } else if (selector == 2) {
    //Secondo for
    sum_diff_uva[(j*maxgrid + i)*length] = diff_uva[(j*maxgrid + i)*length];

    for (int cnt = 1; cnt <= length - 1; cnt++)
      sum_diff_uva[(j*maxgrid + i)*length + cnt] = sum_diff_uva[(j*maxgrid + i)*length + cnt - 1] + diff_uva[(j*maxgrid + i)*length + cnt];
    mean_uva[j*maxgrid + i] = sum_diff_uva[(j*maxgrid + i)*length + (length - 1)];

  } else if (selector == 3 && j == 0) {
    //Terzo for
    path_uva[i] = mean_uva[i];

  } else if (selector == 4 && i == 0 && j == 0) {
    //Quarto for
    for (j = 1; j <= maxgrid - 1; j++)
      for (i = j; i <= maxgrid - 1; i++)
        path_uva[j*maxgrid + i] = path_uva[(j - 1)*maxgrid + (i-1)] + mean_uva[j*maxgrid + i];
  }
}

#define BLOCK_SIZE 32

static void kernel_reg_detect_cuda(int niter, int maxgrid, int length,int* sum_tang_d,int* mean_d,
                              int* path_d, int* diff_d, int* sum_diff_d)
{
  clock_t begin = clock();

  dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
  dim3 dimGrid((maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE,(maxgrid+BLOCK_SIZE-1)/BLOCK_SIZE);

  for (int t = 0; t < niter; t++)
  {   
    // first 3-nested-for
    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 1);

    // second 3-nested-for
    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 2);
    
    // 1-for
    kernel<<<dimGrid, dimBlock>>>(maxgrid, length, sum_tang_d, mean_d, path_d, diff_d, sum_diff_d, 3);

    // sequential part
    hipDeviceSynchronize();
    for (int j = 1; j <= maxgrid - 1; j++)
      for (int i = j; i <= maxgrid - 1; i++)
        path_d[j*maxgrid + i] = path_d[(j - 1)*maxgrid + (i-1)] + mean_d[j*maxgrid + i];
  }

  clock_t end = clock();
  printf("Elapsed time with custom timer: %lf\n", (double)(end - begin) / CLOCKS_PER_SEC);
}

/**
 * Version 3.2
 * Usage of Unified Virtual Memory (UVM)
 * The kernel is a monolythic function with a "multiplexer",
 * in order to choose the correct part of the algorithm.
*/
int main() {
    int niter = NITER;
    int maxgrid = MAXGRID;
    int length = LENGTH;

    int* sum_tang_uva;
    int* sum_diff_uva;
    int* path_uva;
    int* mean_uva;
    int* diff_uva;
    
    hipMallocManaged(&sum_diff_uva, sizeof(int) * maxgrid * maxgrid * length);
    hipMallocManaged(&sum_tang_uva, sizeof(int) * maxgrid * maxgrid);
    hipMallocManaged(&mean_uva, sizeof(int) * maxgrid * maxgrid);
    hipMallocManaged(&path_uva, sizeof(int) * maxgrid * maxgrid);
    hipMallocManaged(&diff_uva, sizeof(int) * maxgrid * maxgrid * length);
    
    init_array(maxgrid, sum_tang_uva, mean_uva, path_uva);

    print_array(maxgrid, path_uva);
    
    kernel_reg_detect_cuda(niter, maxgrid, length, sum_tang_uva, mean_uva, path_uva, diff_uva, sum_diff_uva);
    hipDeviceSynchronize();

    print_array(maxgrid, path_uva);

    hipFree(sum_tang_uva);
    hipFree(sum_diff_uva);
    hipFree(mean_uva);
    hipFree(path_uva);
    hipFree(diff_uva);
    
    return 0;
}
